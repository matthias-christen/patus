#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <cutil.h>

typedef uint64_t gpu_ptr_t;

#pragma patus forward_decls

int main (int argc, char** argv)
{
	int i;
	hipError_t res;

	// prepare grids
	#pragma patus declare_grids
	#pragma patus allocate_grids
	
	#pragma patus declare_GPU_grids
	#pragma patus allocate_GPU_grids
	#pragma patus copy_grids_to_GPU

	#pragma patus initialize_grids
	hipDeviceSynchronize ();
	res = hipGetLastError ();
	if (res != hipSuccess)
	{
		printf ("CUDA Error [Initialization]: %s.\n", hipGetErrorString (res));
		#pragma patus deallocate_grids
		hipDeviceReset ();
		return -1;
	}

	long nFlopsPerStencil = PATUS_FLOPS_PER_STENCIL;
	long nGridPointsCount = 5 * PATUS_GRID_POINTS_COUNT;
	long nBytesTransferred = 5 * PATUS_BYTES_TRANSFERRED;

	// warm up
	#pragma patus compute_stencil
	hipDeviceSynchronize ();
	res = hipGetLastError ();
	if (res != hipSuccess)
	{
		printf ("CUDA Error [Stencil]: %s.\n", hipGetErrorString (res));
		#pragma patus deallocate_grids
		hipDeviceReset ();
		return -1;
	}

	// run the benchmark
	tic ();
	for (i = 0; i < 5; i++)
	{
		#pragma patus compute_stencil
		hipDeviceSynchronize ();
	}
	toc (nFlopsPerStencil, nGridPointsCount, nBytesTransferred);

	// validate
	if (PATUS_DO_VALIDATION)
	{
		#pragma patus initialize_grids
		#pragma patus copy_input_grids_from_GPU_to_reference_grids
		#pragma patus compute_stencil
		#pragma patus copy_output_grids_from_GPU
		#pragma patus validate_computation
		
		if (PATUS_VALIDATES)
			puts ("Validation OK.");
		else
		{
			#pragma patus deallocate_grids
			puts ("Validation failed.");
			hipDeviceReset ();
			return -1;
		}
	}	

	// free memory
	#pragma patus deallocate_grids

	hipDeviceReset ();
	return 0;
}
